#include "hip/hip_runtime.h"
/* ------------
 * This code is provided solely for the personal and private use of
 * students taking the CSC367H5 course at the University of Toronto.
 * Copying for purposes other than this use is expressly prohibited.
 * All forms of distribution of this code, whether as given or with
 * any changes, are expressly prohibited.
 *
 * Authors: Bogdan Simion, Felipe de Azevedo Piovezan
 *
 * All of the files in this directory and all subdirectories are:
 * Copyright (c) 2019 Bogdan Simion
 * -------------
 */

 #include <stdio.h>

 #include "kernels.h"
 #include "clock.h" 


#define MY_MIN(x, y) ((x < y) ? x : y)
#define MAX_BLOCK_SIZE 65535




/*************** COMMON WORK ***********************/
/* Process a single pixel and returns the value of processed pixel
 * TODO: you don't have to implement/use this function, but this is a hint
 * on how to reuse your code.
 * */

void run_kernel2(const int8_t *filter, int32_t dimension, const int32_t *input,
  int32_t *output, int32_t width, int32_t height, float cpu_time) {
  // Figure out how to split the work into threads and call the kernel below.
  /* Allocate memory for the input and result on the kernel */
  int32_t * d_input;
  int32_t * d_output;
  int8_t * d_filter;
  
  float gpu_time, transfer_in, transfer_out;

  hipMalloc((void **)&d_input, width*height*sizeof(int32_t));
  hipMalloc((void **)&d_output, width*height*sizeof(int32_t));
  hipMalloc((void **)&d_filter, 81*sizeof(int8_t));

	Clock clock;
  clock.start();
	/* Copy input from host to device memory */
  hipMemcpy(d_input, input, width*height*sizeof(int32_t), hipMemcpyHostToDevice);
  hipMemcpy(d_output, output, width*height*sizeof(int32_t), hipMemcpyHostToDevice);
  hipMemcpy(d_filter, filter, 81*sizeof(int8_t), hipMemcpyHostToDevice);
  transfer_in = clock.stop();
  

  clock.start();

  
  kernel2<<<height, width>>>(d_filter, dimension, d_input, d_output, width, height);

  unsigned int n; 
  int numThreads, numBlocks, shMemSize, maxBlocks;
  n = width*height;
  maxBlocks = MY_MIN(MAX_BLOCK_SIZE, n);
  numThreads = (n < 1024*2) ? (n>>1) : 1024;
  // Make sure numThreads is a power of 2
  numThreads = check_threads(numThreads);
  numBlocks = MY_MIN(maxBlocks, (n+(numThreads*2-1))/(numThreads*2));
  shMemSize = (numThreads <= 32) ? 2*numThreads * sizeof(int32_t) : numThreads * sizeof(int32_t);

  int32_t * smallest;
  int32_t * largest;


  hipMalloc((void **)&smallest, width*height*sizeof(int32_t));
  hipMalloc((void **)&largest, width*height*sizeof(int32_t));

  
  call_reduce(numThreads, numBlocks, 8*shMemSize, d_output, d_output, largest, smallest, n);



  

  while(numBlocks > 1){
    n = (unsigned int)numBlocks;
    numThreads = (n < 1024*2) ? (n>>1) : 1024;
    // Make sure numThreads is a power of 2
    numThreads = check_threads(numThreads);
    numBlocks = MY_MIN(maxBlocks, (n+(numThreads*2-1))/(numThreads*2));
    shMemSize = (numThreads <= 32) ? 2*numThreads * sizeof(int32_t) : numThreads * sizeof(int32_t);
    call_reduce(numThreads, numBlocks, 8*shMemSize, largest, smallest, largest, smallest, n);
  }
  
  normalize2<<<height, width>>>(d_output, width, height, smallest, largest);

  gpu_time = clock.stop();


  clock.start();
  hipMemcpy(output, d_output, width*height*sizeof(int32_t), hipMemcpyDeviceToHost);
  transfer_out = clock.stop();

  print_run(cpu_time, 2, gpu_time, transfer_in, transfer_out);
  
  hipFree(smallest);
  hipFree(largest);
  hipFree(d_input);
  hipFree(d_output);
  hipFree(d_filter);


}



__global__ void kernel2(const int8_t * filter, int32_t dimension,
                        const int32_t * input, int32_t * output, int32_t width,
                        int32_t height) {
  //sajs
  unsigned int row = blockIdx.x;
  unsigned int col = threadIdx.x;

  // Variable that stores the offset of the pixel we're altering
  int current = row*width + col;    // {0, 1, 2, 3, 4, 5, 6, 7, 8} -> {0, 1, 2,
                                    //                                 3, 4, 5,    
                                    //                                 6, 7, 8}
                                    //
                                    // offset = (row * width_of_matrix) + col

  // Storing value of dimension in easy to reach variable 
  int32_t dim = 9;
  // Variable which will hold the new value for the pixel
  int32_t new_pixel = 0;

  // Variable that stores the offset of the top left corner that the filter matrix covers
  int corner = current - dim/2 - dim/2*width;
  // Variables that store the right and left boundary of the row -> will help when dealing with 'edge' cases hehe
  int start_left = current - dim/2*width - col;
  int start_right = current - dim/2*width + width-col-1;
  int left = start_left;
  int right = start_right;

  // Stores the offset of the current pixel which we are considering
  int curr;
  // Calculating the new pixel value
  for (int dh = 0; dh < dim; dh++){
    for(int dw = 0; dw < dim; dw++){
      curr = corner + dh*width + dw;
      // Check that the current offset is within the boundaries
      if(curr >= left && curr <= right){
        // Check that the offset is within 0 and len(input)
        if(curr >= 0 && curr < width*height){
          new_pixel += input[curr]*filter[dh*dim + dw];
        }
      }
    }
  // Updating the boundaries
  left = left + width;
  right = right + width;
  }

  // Place pixel in output
  output[current] = new_pixel;
    
}

__global__ void normalize2(int32_t *image, int32_t width, int32_t height,
                           int32_t * smallest, int32_t * biggest) {
  unsigned int row = blockIdx.x;
  unsigned int col = threadIdx.x;

  int32_t pixel_idx = row*width + col;

  if (smallest[0] == biggest[0]) {
    return;
  }

  image[pixel_idx] = ((image[pixel_idx] - smallest[0]) * 255) / (biggest[0] - smallest[0]);
               
}


#include "hip/hip_runtime.h"
/* ------------
 * This code is provided solely for the personal and private use of
 * students taking the CSC367H5 course at the University of Toronto.
 * Copying for purposes other than this use is expressly prohibited.
 * All forms of distribution of this code, whether as given or with
 * any changes, are expressly prohibited.
 *
 * Authors: Bogdan Simion, Felipe de Azevedo Piovezan
 *
 * All of the files in this directory and all subdirectories are:
 * Copyright (c) 2019 Bogdan Simion
 * -------------
 */

 #include <stdio.h>

 #include "kernels.h"
 #include "clock.h" 


#define MY_MIN(x, y) ((x < y) ? x : y)
#define MAX_BLOCK_SIZE 65535




/*************** COMMON WORK ***********************/
/* Process a single pixel and returns the value of processed pixel
 * TODO: you don't have to implement/use this function, but this is a hint
 * on how to reuse your code.
 * */

void run_kernel5(const int32_t *input,
  int32_t *output, int32_t width, int32_t height, float cpu_time) {
  // Figure out how to split the work into threads and call the kernel below.
  /* Allocate memory for the input and result on the kernel */
  int32_t * d_input;
  int32_t * d_output;
  
  float gpu_time, transfer_in, transfer_out;

  hipMalloc((void **)&d_input, width*height*sizeof(int32_t));
  hipMalloc((void **)&d_output, width*height*sizeof(int32_t));

	Clock clock;
  clock.start();
	/* Copy input from host to device memory */
  hipMemcpy(d_input, input, width*height*sizeof(int32_t), hipMemcpyHostToDevice);
  transfer_in = clock.stop();
  

  clock.start();

  
  kernel5<<<height, width>>>(d_input, d_output, width, height);

  unsigned int n; 
  int numThreads, numBlocks, shMemSize, maxBlocks;
  n = width*height;
  maxBlocks = MY_MIN(MAX_BLOCK_SIZE, n);
  numThreads = (n < 1024*2) ? (n>>1) : 1024;
  // Make sure numThreads is a power of 2
  numThreads = check_threads(numThreads);
  numBlocks = MY_MIN(maxBlocks, (n+(numThreads*2-1))/(numThreads*2));
  shMemSize = (numThreads <= 32) ? 2*numThreads * sizeof(int32_t) : numThreads * sizeof(int32_t);

  int32_t * smallest;
  int32_t * largest;


  hipMalloc((void **)&smallest, width*height*sizeof(int32_t));
  hipMalloc((void **)&largest, width*height*sizeof(int32_t));

  
  call_reduce(numThreads, numBlocks, 8*shMemSize, d_output, d_output, largest, smallest, n);



  while(numBlocks > 1){
    n = (unsigned int)numBlocks;
    numThreads = (n < 1024*2) ? (n>>1) : 1024;
    // Make sure numThreads is a power of 2
    numThreads = check_threads(numThreads);
    numBlocks = MY_MIN(maxBlocks, (n+(numThreads*2-1))/(numThreads*2));
    shMemSize = (numThreads <= 32) ? 2*numThreads * sizeof(int32_t) : numThreads * sizeof(int32_t);
    call_reduce(numThreads, numBlocks, 8*shMemSize, largest, smallest, largest, smallest, n);
  }

  normalize5<<<height, width>>>(d_output, width, height, smallest, largest);

  gpu_time = clock.stop();


  clock.start();
  hipMemcpy(output, d_output, width*height*sizeof(int32_t), hipMemcpyDeviceToHost);
  transfer_out = clock.stop();

  print_run(cpu_time, 5, gpu_time, transfer_in, transfer_out);
  
  hipFree(smallest);
  hipFree(largest);
  hipFree(d_input);
  hipFree(d_output);


}



__global__ void kernel5(const int32_t * input, int32_t * output, int32_t width,
                        int32_t height) {
  //sajs

  const int8_t filter[] = {
    0, 1, 1, 2, 2, 2,   1,   1,   0, 1, 2, 4, 5, 5,   5,   4,   2,
    1, 1, 4, 5, 3, 0,   3,   5,   4, 1, 2, 5, 3, -12, -24, -12, 3,
    5, 2, 2, 5, 0, -24, -40, -24, 0, 5, 2, 2, 5, 3,   -12, -24, -12,
    3, 5, 2, 1, 4, 5,   3,   0,   3, 5, 4, 1, 1, 2,   4,   5,   5,
    5, 4, 2, 1, 0, 1,   1,   2,   2, 2, 1, 1, 0,
  };
  unsigned int row = blockIdx.x;
  unsigned int col = threadIdx.x;

  // Variable that stores the offset of the pixel we're altering
  int current = row*width + col;    // {0, 1, 2, 3, 4, 5, 6, 7, 8} -> {0, 1, 2,
                                    //                                 3, 4, 5,    
                                    //                                 6, 7, 8}
                                    //
                                    // offset = (row * width_of_matrix) + col

  current = (current + blockDim.x)%(width*height); // Striding
  


  // Storing value of dimension in easy to reach variable 
  int32_t dim = 9;
  // Variable which will hold the new value for the pixel
  int32_t new_pixel = 0;

  // Variable that stores the offset of the top left corner that the filter matrix covers
  int corner = current - dim/2 - dim/2*width;
  // Variables that store the right and left boundary of the row -> will help when dealing with 'edge' cases hehe
  int start_left = current - dim/2*width - col;
  int start_right = current - dim/2*width + width-col-1;
  int left = start_left;
  int right = start_right;

  // Stores the offset of the current pixel which we are considering
  int curr;
  // Calculating the new pixel value
  if (row < 9 || height - row < 9 || col < 9 || width - col < 9  ){
    for (int dh = 0; dh < dim; dh++){
      for(int dw = 0; dw < dim; dw++){
        curr = corner + dh*width + dw;
        // Check that the current offset is within the boundaries
        if(curr >= left && curr <= right){
          // Check that the offset is within 0 and len(input)
          if(curr >= 0 && curr < width*height){
            new_pixel += input[curr]*filter[dh*dim + dw];
          }
        }
      }
    // Updating the boundaries
    left = left + width;
    right = right + width;
    }
  }

  // Place pixel in output
  output[current] = new_pixel;
    
}

__global__ void normalize5(int32_t *image, int32_t width, int32_t height,
                           int32_t * smallest, int32_t * biggest) {
  unsigned int row = blockIdx.x;
  unsigned int col = threadIdx.x;

  int32_t pixel_idx = row*width + col;

  if (smallest[0] == biggest[0]) {
    return;
  }

  image[pixel_idx] = ((image[pixel_idx] - smallest[0]) * 255) / (biggest[0] - smallest[0]);
               
}



/* ------------
 * This code is made by two university of toronto students, but was inspired by code that's property 
 * of the University of Toronto. 
 * We don't care abour copyright, however this makes our code look more professional :^)
 *
 * Authors: Miguel De Vera, Mark Sedhom
 * Inspiration: Bogdan Simion, Felipe de Azevedo Piovezan
 *
 * -------------
 */


#include "kernels.h"
#include "stdio.h"

#define MAX_BLOCK_SIZE 65535

void call_reduce(int numThreads, int numBlocks, int shMemSize, int32_t * d_output1, int32_t * d_output2, 
    int32_t * largest, int32_t * smallest, unsigned int n){
  switch (numThreads) {
    case 512:
    reduce_kernel<512><<<numBlocks, numThreads, shMemSize>>>(d_output1, d_output2, largest, smallest, n);
    break;
    case 256:
    reduce_kernel<256><<<numBlocks, numThreads, shMemSize>>>(d_output1, d_output2, largest, smallest, n);
    break;
    case 128:
    reduce_kernel<128><<<numBlocks, numThreads, shMemSize>>>(d_output1, d_output2, largest, smallest, n);
    break;
    case 64:
    reduce_kernel<64><<<numBlocks, numThreads, shMemSize>>>(d_output1, d_output2, largest, smallest, n);
    break;
    case 32:
    reduce_kernel<32><<<numBlocks, numThreads, shMemSize>>>(d_output1, d_output2, largest, smallest, n);
    break;
    case 16:
    reduce_kernel<16><<<numBlocks, numThreads, shMemSize>>>(d_output1, d_output2, largest, smallest, n);
    break;
    case 8:
    reduce_kernel<8><<<numBlocks, numThreads, shMemSize>>>(d_output1, d_output2, largest, smallest, n);
    break;
    case 4:
    reduce_kernel<4><<<numBlocks, numThreads, shMemSize>>>(d_output1, d_output2, largest, smallest, n);
    break;
    case 2:
    reduce_kernel<2><<<numBlocks, numThreads, shMemSize>>>(d_output1, d_output2, largest, smallest, n);
    break;
    case 1:
    reduce_kernel<1><<<numBlocks, numThreads, shMemSize>>>(d_output1, d_output2, largest, smallest, n);
    break;
    default:
    printf("invalid number of threads, exiting...\n");
    exit(1);
  }
}

int check_threads(int numThreads){
  if (numThreads > 512){
  return 512;
  }else if (numThreads > 256){
  return 256;
  }else if (numThreads > 128){
  return 128;
  }else if (numThreads > 64){
  return 64;
  }else if (numThreads > 32){
  return 32;
  }else if (numThreads > 16){
  return 16;
  }else if (numThreads > 8){
  return 8;
  }else if (numThreads > 4){
  return 4;
  }else if (numThreads > 2){
  return 2;
  }else{
  return 1;
  }
}


//supposed to be void - changed for debugging
__device__ void apply2d(const int8_t *filter, int32_t dimension,
  const int32_t *input, int32_t *output, int32_t width,
  int32_t height, int row, int col){
  // Variable that stores the offset of the pixel we're altering
  int current = row*width + col;    // {0, 1, 2, 3, 4, 5, 6, 7, 8} -> {0, 1, 2,
                                    //                                 3, 4, 5,    
                                    //                                 6, 7, 8}
                                    //
                                    // offset = (row * width_of_matrix) + col

  // Storing value of dimension in easy to reach variable 
  int32_t dim = 9;
  // Variable which will hold the new value for the pixel
  int32_t new_pixel = 0;

  // Variable that stores the offset of the top left corner that the filter matrix covers
  int corner = current - dim/2 - dim/2*width;
  // Variables that store the right and left boundary of the row -> will help when dealing with 'edge' cases hehe
  int start_left = current - dim/2*width - col;
  int start_right = current - dim/2*width + width-col-1;
  int left = start_left;
  int right = start_right;

  // Stores the offset of the current pixel which we are considering
  int curr;
  // Calculating the new pixel value
  for (int dh = 0; dh < dim; dh++){
  for(int dw = 0; dw < dim; dw++){
  curr = corner + dh*width + dw;
  // Check that the current offset is within the boundaries
  if(curr >= left && curr <= right){
  // Check that the offset is within 0 and len(input)
  if(curr >= 0 && curr < width*height){
  new_pixel += input[curr]*filter[dh*dim + dw];
  }
  }
  }
  // Updating the boundaries
  left = left + width;
  right = right + width;
  }

  // Place pixel in output
  output[current] = new_pixel;
  // return new_pixel;
}
  
  
__device__ void normalize_one(int32_t *image, int32_t width, int32_t height,
  int32_t * smallest, int32_t * biggest, int row, int col){

  int32_t pixel_idx = row*width + col;

  if (smallest[0] == biggest[0]) {
    return;
  }

  image[pixel_idx] = ((image[pixel_idx] - smallest[0]) * 255) / (biggest[0] - smallest[0]);
}


/* Use this function to print the time of each of your kernels.
* The parameter names are intuitive, but don't hesitate to ask
* for clarifications.
* DO NOT modify this function.*/
void print_run(float time_cpu, int kernel, float time_gpu_computation,
  float time_gpu_transfer_in, float time_gpu_transfer_out) {
printf("%12.6f ", time_cpu);
printf("%5d ", kernel);
printf("%12.6f ", time_gpu_computation);
printf("%14.6f ", time_gpu_transfer_in);
printf("%15.6f ", time_gpu_transfer_out);
printf("%13.2f ", time_cpu / time_gpu_computation);
printf("%7.2f\n", time_cpu / (time_gpu_computation + time_gpu_transfer_in +
                  time_gpu_transfer_out));
}
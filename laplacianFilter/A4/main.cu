#include "hip/hip_runtime.h"
/* ------------
 * This code is provided solely for the personal and private use of
 * students taking the CSC367H5 course at the University of Toronto.
 * Copying for purposes other than this use is expressly prohibited.
 * All forms of distribution of this code, whether as given or with
 * any changes, are expressly prohibited.
 *
 * Authors: Bogdan Simion, Felipe de Azevedo Piovezan
 *
 * All of the files in this directory and all subdirectories are:
 * Copyright (c) 2019 Bogdan Simion
 * -------------
 */

#include <stdio.h>
#include <string>
#include <unistd.h>

#include "pgm.h"
#include "clock.h"
#include "kernels.h"


int main(int argc, char **argv) {
  int c;
  std::string input_filename, cpu_output_filename, base_gpu_output_filename;
  if (argc < 3) {
    printf("Wrong usage. Expected -i <input_file> -o <output_file>\n");
    return 0;
  }

  while ((c = getopt(argc, argv, "i:o:")) != -1) {
    switch (c) {
    case 'i':
      input_filename = std::string(optarg);
      break;
    case 'o':
      cpu_output_filename = std::string(optarg);
      base_gpu_output_filename = std::string(optarg);
      break;
    default:
      return 0;
    }
  }

  pgm_image source_img;
  init_pgm_image(&source_img);
  float cpu_time = 0.0;
  const int8_t FILTER[] = {
    0, 1, 1, 2, 2, 2,   1,   1,   0, 1, 2, 4, 5, 5,   5,   4,   2,
    1, 1, 4, 5, 3, 0,   3,   5,   4, 1, 2, 5, 3, -12, -24, -12, 3,
    5, 2, 2, 5, 0, -24, -40, -24, 0, 5, 2, 2, 5, 3,   -12, -24, -12,
    3, 5, 2, 1, 4, 5,   3,   0,   3, 5, 4, 1, 1, 2,   4,   5,   5,
    5, 4, 2, 1, 0, 1,   1,   2,   2, 2, 1, 1, 0,
  };

  const int FILTER_DIMENSION = 9;

  if (load_pgm_from_file(input_filename.c_str(), &source_img) != NO_ERR) {
    printf("Error loading source image.\n");
    return 0;
  }

  /* Do not modify this printf */
  printf("CPU_time(ms) Kernel GPU_time(ms) TransferIn(ms) TransferOut(ms) "
         "Speedup_noTrf Speedup\n");

  /* TODO: run your CPU implementation here and get its time. Don't include
   * file IO in your measurement.*/
  /* For example: */
  {
    std::string cpu_file = cpu_output_filename;
    pgm_image cpu_output_img;
    copy_pgm_image_size(&source_img, &cpu_output_img);
    // Start time
    // run_best_cpu(args...);  // From kernels.h
    // End time
    // print_run(args...)      // Defined on the top of this file

    Clock clock; 
    clock.start();
    run_best_cpu(FILTER, FILTER_DIMENSION, source_img.matrix, cpu_output_img.matrix, source_img.width,
      source_img.height);
    cpu_time = clock.stop();    

    save_pgm_to_file(cpu_file.c_str(), &cpu_output_img);
    destroy_pgm_image(&cpu_output_img);
  }

  /* TODO:
   * run each of your gpu implementations here,
   * get their time,
   * and save the output image to a file.
   * Don't forget to add the number of the kernel
   * as a prefix to the output filename:
   * Print the execution times by calling print_run().
   */

  /* For example: */
  {
    std::string gpu_file = "1" + base_gpu_output_filename;
    pgm_image gpu_output_img;
    copy_pgm_image_size(&source_img, &gpu_output_img);
    // Start time
    // run_kernel1(args...);  // From kernels.h
    // End time
    // print_run(args...)     // Defined on the top of this file
    Clock clock; 
    clock.start();
    run_kernel1(FILTER, FILTER_DIMENSION, source_img.matrix, gpu_output_img.matrix, source_img.width,
      source_img.height, cpu_time);

    save_pgm_to_file(gpu_file.c_str(), &gpu_output_img);
    destroy_pgm_image(&gpu_output_img);
  }

  {
    std::string gpu_file = "2" + base_gpu_output_filename;
    pgm_image gpu_output_img;
    copy_pgm_image_size(&source_img, &gpu_output_img);
    // Start time
    // run_kernel1(args...);  // From kernels.h
    // End time
    // print_run(args...)     // Defined on the top of this file

    run_kernel2(FILTER, FILTER_DIMENSION, source_img.matrix, gpu_output_img.matrix, source_img.width,
      source_img.height, cpu_time);

    save_pgm_to_file(gpu_file.c_str(), &gpu_output_img);
    destroy_pgm_image(&gpu_output_img);
  }

  {
    std::string gpu_file = "3" + base_gpu_output_filename;
    pgm_image gpu_output_img;
    copy_pgm_image_size(&source_img, &gpu_output_img);
    // Start time
    // run_kernel1(args...);  // From kernels.h
    // End time
    // print_run(args...)     // Defined on the top of this file
    Clock clock; 
    clock.start();
    run_kernel3(FILTER, FILTER_DIMENSION, source_img.matrix, gpu_output_img.matrix, source_img.width,
      source_img.height, cpu_time);

    save_pgm_to_file(gpu_file.c_str(), &gpu_output_img);
    destroy_pgm_image(&gpu_output_img);
  }

  {
    std::string gpu_file = "4" + base_gpu_output_filename;
    pgm_image gpu_output_img;
    copy_pgm_image_size(&source_img, &gpu_output_img);
    // Start time
    // run_kernel1(args...);  // From kernels.h
    // End time
    // print_run(args...)     // Defined on the top of this file
    Clock clock; 
    clock.start();
    run_kernel4(FILTER, FILTER_DIMENSION, source_img.matrix, gpu_output_img.matrix, source_img.width,
      source_img.height, cpu_time);

    save_pgm_to_file(gpu_file.c_str(), &gpu_output_img);
    destroy_pgm_image(&gpu_output_img);
  }

  {
    std::string gpu_file = "5" + base_gpu_output_filename;
    pgm_image gpu_output_img;
    copy_pgm_image_size(&source_img, &gpu_output_img);
    // Start time
    // run_kernel1(args...);  // From kernels.h
    // End time
    // print_run(args...)     // Defined on the top of this file

    run_kernel5(source_img.matrix, gpu_output_img.matrix, source_img.width,
      source_img.height, cpu_time);

    save_pgm_to_file(gpu_file.c_str(), &gpu_output_img);
    destroy_pgm_image(&gpu_output_img);
  }

  /* Repeat that for all 5 kernels. Don't hesitate to ask if you don't
   * understand the idea. */
}

/* ------------
 * This code is provided solely for the personal and private use of
 * students taking the CSC367H5 course at the University of Toronto.
 * Copying for purposes other than this use is expressly prohibited.
 * All forms of distribution of this code, whether as given or with
 * any changes, are expressly prohibited.
 *
 * Authors: Bogdan Simion, Felipe de Azevedo Piovezan
 *
 * All of the files in this directory and all subdirectories are:
 * Copyright (c) 2019 Bogdan Simion
 * -------------
 */

#include "kernels.h"
#include <pthread.h>
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>

pthread_mutex_t mutex;

typedef struct common_work_t
{
    const int8_t *f;
    const int32_t *original_image;
    int32_t *output_image;
    int32_t width;
    int32_t height;
    int32_t max_threads;
    int32_t smallest;
    int32_t largest;
    pthread_barrier_t * barrier;
} common_work;

typedef struct work_t
{
    common_work *common;
    int32_t id;
} work_t;



/* Normalizes a pixel given the smallest and largest integer values
 * in the image */
void normalize_pixelcpu(int32_t *target, int32_t pixel_idx, int32_t smallest,
    int32_t largest) {
    if (smallest == largest) {
        return;
    }

    target[pixel_idx] = ((target[pixel_idx] - smallest) * 255) / (largest - smallest);
}




int32_t apply2dcpu(const int8_t *f, const int32_t *original, int32_t *target,
                int32_t width, int32_t height, int row, int column) {

    // Variable that stores the offset of the pixel we're altering
    int current = row*width + column;     // {0, 1, 2, 3, 4, 5, 6, 7, 8} -> {0, 1, 2,
                                          //                                 3, 4, 5,    
                                          //                                 6, 7, 8}
                                          //
                                          // offset = (row * width_of_matrix) + column

    // Storing value of dimension in easy to reach variable 
    int32_t dim = 9;
    // Variable which will hold the new value for the pixel
    int32_t new_pixel = 0;

    // Variable that stores the offset of the top left corner that the filter matrix covers
    int corner = current - dim/2 - dim/2*width;
    // Variables that store the right and left boundary of the row -> will help when dealing with 'edge' cases hehe
    int start_left = current - dim/2*width - column;
    int start_right = current - dim/2*width + width-column-1;
    int left = start_left;
    int right = start_right;

    // Stores the offset of the current pixel which we are considering
    int curr;
    // Calculating the new pixel value
    for (int dh = 0; dh < dim; dh++){
        for(int dw = 0; dw < dim; dw++){

            curr = corner + dh*width + dw;
            // Check that the current offset is within the boundaries
            if(curr >= left && curr <= right){
                // Check that the offset is within 0 and len(original)
                if(curr >= 0 && curr < width*height){
                    new_pixel += original[curr]*f[dh*dim + dw];
                }
            }
        }
        // Updating the boundaries
        left = left + width;
        right = right + width;
    }

    // Place pixel in target
    target[current] = new_pixel;

    return new_pixel;
}



void *sharding_work(void *work) {
    /* Your algorithm is essentially:
     *  1- Apply the filter on the image
     *  2- Wait for all threads to do the same
     *  3- Calculate global smallest/largest elements on the resulting image
     *  4- Scale back the pixels of the image. For the non work queue
     *      implementations, each thread should scale the same pixels
     *      that it worked on step 1.
     */
  
    
    
    work_t * not_common = (work_t*)work;
  
    // Extracting non shared data
    common_work * common = not_common->common;
    int32_t id = not_common->id;
    
    // Extracting all shared data
    pthread_mutex_lock(&mutex);
    int32_t height = common->height;
    int32_t width = common->width;
    int32_t num_threads = common->max_threads;
    const int8_t * f = common->f;
    const int32_t * original = common->original_image;
    int32_t * target = common->output_image;
    pthread_barrier_t * barrier = common->barrier;
    int32_t smallest = common->smallest;
    int32_t largest = common->largest;
    pthread_mutex_unlock(&mutex);
  
    int start, end;
    
    int rows_amount; 
    // Calculating the number of rows we will apply the filter to
    if (height%num_threads == 0){
        rows_amount = height/num_threads;
    }else{
        rows_amount = (height + num_threads - 1)/num_threads;
    }
    // Calculating the offset of where this thread starts altering and ends
    start = rows_amount * id*width;
    end = start + rows_amount*width; 
    if (start > width*height){
        start = width*height;
    }
    if (end > width*height){
        end = width*height;
    }
    for(int offset = start; offset<end; offset++){
        // Modifying the pixel
        int32_t px = apply2dcpu(f, original, target, width, height, offset/width, offset%width);
        // Checking for local smallest
        if(smallest > px){
            smallest = px;
        }
        // Checking for local largest
        if (largest < px){
            largest = px;
        }
    }

    pthread_mutex_lock(&mutex);
    // Checking for global smallest
    if(common->smallest > smallest){
        common->smallest = smallest;
    }
    // Checking for global largest
    if (common->largest < largest){
        common->largest = largest;
    }

    pthread_mutex_unlock(&mutex);

    pthread_barrier_wait(barrier);
    
    pthread_mutex_lock(&mutex);
    // Making sure local largest and smallest is updated
    smallest = common->smallest;
    largest = common->largest;
    pthread_mutex_unlock(&mutex);
    
    // Normalizing all target pixels
    
    for(int offset = start; offset<end; offset++){
        // printf("output[%d]:%d,   smallest:%d, largest:%d\n", offset, target[offset], smallest, largest);
        normalize_pixelcpu(target, offset, smallest, largest);
    }
    
  
    
    
    pthread_exit(NULL);
    return NULL;
  
  
}






void run_best_cpu(const int8_t *filter, int32_t dimension, const int32_t *input,
                  int32_t *output, int32_t width, int32_t height) {
    
                    
    int32_t num_threads = height;
    pthread_t threads[num_threads];

    // The barrier and mutex that will help keep threads in sync
    pthread_barrier_t barrier;
    pthread_barrier_init(&barrier, NULL, num_threads);
    pthread_mutex_init(&mutex, NULL);
    
    common_work * common = (common_work*)malloc(sizeof(common_work));
    *common = {filter, input, output, width, height, height, INT_MAX, INT_MIN, &barrier};
    
    work_t * not_common = (work_t*)malloc(num_threads*sizeof(work_t)); 
    
    for(int t = 0; t<num_threads; t++){
        not_common[t] = {common, t};
        if (pthread_create(&threads[t], NULL, sharding_work, (void*)&not_common[t])){
            fprintf(stderr, "Error with creating a pthread");
            exit(1);
        }
    }
    
    // Waiting for all threads to finish
    for(int i = 0; i<num_threads; i++){
        pthread_join(threads[i], NULL);
    }
    
    free(common);
    free(not_common);


}
